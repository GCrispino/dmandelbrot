#include <iostream>
#include <string>
#include <png++/image.hpp>
#include <mpi.h>

#define QUOTEME(x) QUOTEME_1(x)
#define QUOTEME_1(x) #x
#ifdef __HIPCC__
#   define  __CUDA__ 1
#   define INCLUDE_FILE(x) QUOTEME(thrust/complex.h)
#   define COMPLEX thrust
#else
#   define  __CUDA__ 0
#   define INCLUDE_FILE(x) QUOTEME(complex)
#   define COMPLEX std
#endif

#ifndef REAL_TYPE
#   define REAL_TYPE float
#endif

#include INCLUDE_FILE()

#include "mandelbrot.cuh"


void print_table(unsigned w, unsigned h, unsigned ** table){
    for (unsigned i = 0;i < h; ++i){
        for (unsigned j = 0;j < w; ++j){
            std::cout << table[i * h + j]  << ' ';
        }

        std::cout << std::endl;
    }
}

png::image<png::rgb_pixel> create_image(unsigned w, unsigned h, unsigned *table){

    png::image< png::rgb_pixel > image(w, h);

    #pragma omp parallel for
    for (png::uint_32 y = 0; y < image.get_height(); ++y)
    {
       for (png::uint_32 x = 0; x < image.get_width(); ++x)
       {
            if (table[y * w + x] == 0){
                image[y][x] = png::rgb_pixel(30, 30, 30);
            }
            else{
                image[y][x] = png::rgb_pixel(table[y * w + x] * 2, table[y * w + x] * 2, 170 + table[y * w + x] * 2);
            }
        }
    }

    return image;
}


struct params {
    COMPLEX::complex<REAL_TYPE> c0,c1;
    unsigned w,h,n_threads;
    mandelbrot::exec_mode ex;
    std::string output_path;

    params(
        const COMPLEX::complex<REAL_TYPE> &c0, const COMPLEX::complex<REAL_TYPE> &c1,
        unsigned w, unsigned h, unsigned n_threads,
        mandelbrot::exec_mode ex, const std::string &output_path
    ): c0(c0), c1(c1), w(w), h(h), n_threads(n_threads), ex(ex), output_path(output_path)
    {}
};

mandelbrot::exec_mode get_exec_mode(const char * mode){
    mandelbrot::exec_mode ex;
    if (mode[0] == '0'){
        ex = mandelbrot::exec_mode::CPU;
    }
    else if (mode[0] == '1'){
        if (!__CUDA__){
            std::cerr << "WARNING! You chose to use GPU execution without using nvcc" << std::endl;
            std::cerr << "\tDefaulting to CPU execution..." << std::endl;
            return mandelbrot::exec_mode::CPU;
        }
        ex = mandelbrot::exec_mode::GPU;
    }
    else{
        std::cerr << "Invalid execution mode (0 or 1 is allowed)!" << std::endl;
        exit(1);
    }

    return ex;
}

struct params parse_args(int argc, char **argv){

    using COMPLEX::complex;


    std::string
        usage("USAGE: dmbrot <C0_REAL_TYPE> <C0_IMAG> <C1_REAL> <C1_IMAG> <W> <H> <CPU/GPU> <THREADS> <OUTPUT>");

    if (argc != 10){
        std::cerr << usage << std::endl;
        exit(1);
    }

    REAL_TYPE
        c0_real = atof(argv[1]), c0_imag = atof(argv[2]),
        c1_real = atof(argv[3]), c1_imag = atof(argv[4]);

    unsigned w = atoi(argv[5]), h = atoi(argv[6]), n_threads = atoi(argv[8]);

    const complex<REAL_TYPE> c0(c0_real, c0_imag), c1(c1_real, c1_imag);

    return params(
        c0,c1,
        w, h, n_threads, get_exec_mode(argv[7]),argv[9]
    );
}


int main(int argc, char **argv){
    using COMPLEX::complex;

    MPI_Init(&argc, &argv);

    int world_size, world_rank;

    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    params args = parse_args(argc, argv);

    const mandelbrot::exec_mode ex = args.ex;
    const unsigned w = args.w, h = args.h, m = 250;

    complex<REAL_TYPE> c0(args.c0),c1(args.c1);

    const REAL_TYPE delta_x = (c1.real() - c0.real()) / w;
    const REAL_TYPE delta_y = (c1.imag() - c0.imag()) / h;

    if(world_size > h + 1){
        if (world_rank == 0){
            std::cerr << "Number of processes cannot be higher than image height + 1!" << std::endl;
            std::cerr << "Setting number of processes to h + 1 = " << h + 1 << std::endl;
        }

        world_size = h + 1; 
    }

    world_size = world_size > h + 1 ? h + 1 : world_size;

    if (world_size == 1){
        if (world_rank == 0){
            std::cerr << "Number of processes needs to be at least 2! Exiting..." << std::endl;
        }

        MPI_Finalize();
        exit(1);
    }

    if (world_rank > world_size - 1){
        MPI_Finalize();
        exit(0);
    }

    if (world_rank == 0){
        std::cout << "Exec mode: " << ex << std::endl;

        std::cout << "c0: (" << c0.real() << ',' << c0.imag() << ")" << std::endl;
        std::cout << "c1: (" << c1.real() << ',' << c1.imag() << ")" << std::endl;
        std::cout << "w: " << w << ", h: " << h << std::endl;
        std::cout << "Delta x: " << delta_x << std::endl;
        std::cout << "Delta y: " << delta_y << std::endl;

        unsigned *table = new unsigned[w * h];
        unsigned block_size = mandelbrot::get_block_size(w, h, world_size - 1);

        for (unsigned i = 1; i < world_size; ++i){
            int *bound = mandelbrot::get_boundaries(w, h, i, world_size - 1, block_size);
            int start_x = bound[0], end_x = bound[1], start_y = bound[2], end_y = bound[3];
            unsigned local_h = end_y - start_y + 1;
            unsigned local_table_size = w * local_h;
            unsigned offset = block_size * w * (i - 1);

            MPI_Recv(
                table + offset,
                local_table_size,
                MPI_UNSIGNED,
                i,
                0,
                MPI_COMM_WORLD,
                MPI_STATUS_IGNORE
            );
        }
            
        png::image< png::rgb_pixel > image = create_image(w,h,table);
        image.write(args.output_path);

        delete[] table;
    }
    else{
        d_mandelbrot(
            ex, args.n_threads,
            world_rank, world_size - 1,
            c0, c1,
            delta_x, delta_y,
            w, h, m
        );
    }


    MPI_Finalize();

    return 0;
}
