#include <iostream>
#include <string>
#include <functional>
#include <png++/image.hpp>
#include <mpi.h>

#define QUOTEME(x) QUOTEME_1(x)
#define QUOTEME_1(x) #x
#ifdef __HIPCC__
#   define  __CUDA__ 1
#   define INCLUDE_FILE(x) QUOTEME(thrust/complex.h)
#   define COMPLEX thrust
#else
#   define  __CUDA__ 0
#   define INCLUDE_FILE(x) QUOTEME(complex)
#   define COMPLEX std
#endif

#ifndef REAL_TYPE
#   define REAL_TYPE float
#endif

#include INCLUDE_FILE()

#include "mandelbrot.cuh"


png::image<png::rgb_pixel> create_image(unsigned w, unsigned h, unsigned *table){

    png::image< png::rgb_pixel > image(w, h);

    #pragma omp parallel for
    for (png::uint_32 y = 0; y < image.get_height(); ++y)
    {
       for (png::uint_32 x = 0; x < image.get_width(); ++x)
       {
            if (table[y * w + x] == 0){
                image[y][x] = png::rgb_pixel(30, 30, 30);
            }
            else{
                image[y][x] = png::rgb_pixel(table[y * w + x] * 2, table[y * w + x] * 2, 170 + table[y * w + x] * 2);
            }
        }
    }

    return image;
}


struct params {
    COMPLEX::complex<REAL_TYPE> c0,c1;
    unsigned w,h,n_threads;
    mandelbrot::exec_mode ex;
    std::string output_path;

    params(
        const COMPLEX::complex<REAL_TYPE> &c0, const COMPLEX::complex<REAL_TYPE> &c1,
        unsigned w, unsigned h, unsigned n_threads,
        mandelbrot::exec_mode ex, const std::string &output_path
    ): c0(c0), c1(c1), w(w), h(h), n_threads(n_threads), ex(ex), output_path(output_path)
    {}
};

mandelbrot::exec_mode get_exec_mode(const char * mode){
    mandelbrot::exec_mode ex;
    if (mode[0] == '0'){
        ex = mandelbrot::exec_mode::CPU;
    }
    else if (mode[0] == '1'){
        if (!__CUDA__){
            std::cerr << "WARNING! You chose to use GPU execution without using nvcc" << std::endl;
            std::cerr << "\tDefaulting to CPU execution..." << std::endl;
            return mandelbrot::exec_mode::CPU;
        }
        ex = mandelbrot::exec_mode::GPU;
    }
    else{
        std::cerr << "Invalid execution mode (0 or 1 is allowed)!" << std::endl;
        exit(1);
    }

    return ex;
}

struct params parse_args(int argc, char **argv){

    using COMPLEX::complex;


    std::string
        usage("USAGE: dmbrot <C0_REAL_TYPE> <C0_IMAG> <C1_REAL> <C1_IMAG> <W> <H> <CPU/GPU> <THREADS> <OUTPUT>");

    if (argc != 10){
        std::cerr << usage << std::endl;
        exit(1);
    }

    REAL_TYPE
        c0_real = atof(argv[1]), c0_imag = atof(argv[2]),
        c1_real = atof(argv[3]), c1_imag = atof(argv[4]);

    unsigned w = atoi(argv[5]), h = atoi(argv[6]), n_threads = atoi(argv[8]);

    const complex<REAL_TYPE> c0(c0_real, c0_imag), c1(c1_real, c1_imag);

    return params(
        c0,c1,
        w, h, n_threads, get_exec_mode(argv[7]),argv[9]
    );
}


template <class R, class ...A>
R do_master(int rank, std::function<R(A...)> fn, std::function<R(A...)> fn_else, A... args){
    if (rank == 0){ // master
        return fn(args...);
    }

    return fn_else(args...); // slave
}

void master_work(
    params args,
    int rank, int n_procs,
    REAL_TYPE delta_x, REAL_TYPE delta_y,
    unsigned m
){
    COMPLEX::complex<REAL_TYPE> c0(args.c0),c1(args.c1);
    const unsigned w = args.w, h = args.h;
    std::cout << "Exec mode: " << args.ex << std::endl;

    std::cout << "c0: (" << c0.real() << ',' << c0.imag() << ")" << std::endl;
    std::cout << "c1: (" << c1.real() << ',' << c1.imag() << ")" << std::endl;
    std::cout << "w: " << w << ", h: " << h << std::endl;
    std::cout << "Delta x: " << delta_x << std::endl;
    std::cout << "Delta y: " << delta_y << std::endl;

    unsigned *table = new unsigned[w * h];
    unsigned block_size = mandelbrot::get_block_size(w, h, n_procs - 1);

    for (unsigned i = 1; i < n_procs; ++i){
        int *bound = mandelbrot::get_boundaries(w, h, i, n_procs - 1, block_size);
        int start_y = bound[2], end_y = bound[3];
        unsigned local_h = end_y - start_y + 1;
        unsigned local_table_size = w * local_h;
        unsigned offset = block_size * w * (i - 1);

        MPI_Recv(
            table + offset,
            local_table_size,
            MPI_UNSIGNED,
            i,
            0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }
        
    png::image< png::rgb_pixel > image = create_image(w,h,table);
    image.write(args.output_path);

    delete[] table;
}


void slave_work(
    params args,
    int rank, int n_procs,
    REAL_TYPE delta_x, REAL_TYPE delta_y,
    unsigned m
){
    d_mandelbrot(
        args.ex, args.n_threads,
        rank, n_procs - 1,
        args.c0, args.c1,
        delta_x, delta_y,
        args.w, args.h, m
    );
}

int main(int argc, char **argv){
    using COMPLEX::complex;
    using std::function;

    int error;

    // MPI Initialization
    // ----------------------------------------------
    if (error = MPI_Init(&argc, &argv)){
        std::cout << "Error in MPI_Init: " << error << std::endl;
    }


    int world_size, world_rank;
    if (error = MPI_Comm_size(MPI_COMM_WORLD, &world_size)){
        std::cout << "Error in MPI_Comm_size: " << error << std::endl;
    }

    if (error = MPI_Comm_rank(MPI_COMM_WORLD, &world_rank)){
        std::cout << "Error in MPI_Comm_rank: " << error << std::endl;
    }
    // ----------------------------------------------

    params args = parse_args(argc, argv);

    unsigned w = args.w, h = args.h, m = 250;

    complex<REAL_TYPE> c0(args.c0),c1(args.c1);

    REAL_TYPE delta_x = (c1.real() - c0.real()) / w;
    REAL_TYPE delta_y = (c1.imag() - c0.imag()) / h;

    function <void(unsigned)>dummy = [](unsigned){};

    if(world_size > h + 1){
        function<void(unsigned)> proc_number_err_fn = function<void(unsigned)>(
            [](unsigned h){
                std::cerr << "Number of processes cannot be higher than image height + 1!" << std::endl;
                std::cerr << "Setting number of processes to h + 1 = " << h + 1 << std::endl;
            }
        );

        do_master(
            world_rank,
            proc_number_err_fn,
            dummy, h
        );

        world_size = h + 1; 
    }

    world_size = world_size > h + 1 ? h + 1 : world_size;

    if (world_size == 1){
        function<void(unsigned)> proc_number_err_fn = function<void(unsigned)>(
            [](unsigned h){
                std::cerr << "Number of processes needs to be at least 2! Exiting..." << std::endl;
            }
        );

        do_master(
            world_rank,
            proc_number_err_fn,
            dummy, h
        );


        if (error = MPI_Finalize()){
            std::cout << "Error in MPI_Finalize: " << error << std::endl;
        }

        exit(1);
    }

    if (world_rank > world_size - 1){
        if (error = MPI_Finalize()){
            std::cout << "Error in MPI_Finalize: " << error << std::endl;
        }
        exit(0);
    }
    
    std::function<void(
        params, int, int,
        REAL_TYPE, REAL_TYPE, unsigned
    )> slave_fn = slave_work, master_fn = master_work;

    // do master and slave main work depending on rank
    do_master(
        world_rank,
        master_fn,
        slave_fn,
        args, world_rank, world_size,
        delta_x, delta_y, m
    );

    if (error = MPI_Finalize()){
        std::cout << "Error in MPI_Finalize: " << error << std::endl;
    }

    return 0;
}
