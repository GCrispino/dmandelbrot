#include <iostream>
#include <string>
#include <png++/image.hpp>
#include <mpi.h>

#define QUOTEME(x) QUOTEME_1(x)
#define QUOTEME_1(x) #x
#ifdef __HIPCC__
#   define  __CUDA__ 1
#   define INCLUDE_FILE(x) QUOTEME(thrust/complex.h)
#   define COMPLEX thrust
#else
#   define  __CUDA__ 0
#   define INCLUDE_FILE(x) QUOTEME(complex)
#   define COMPLEX std
#endif

#ifndef REAL_TYPE
#   define REAL_TYPE float
#endif

#include INCLUDE_FILE()

#include "mandelbrot.cuh"


void print_table(unsigned w, unsigned h, unsigned ** table){
    for (unsigned i = 0;i < h; ++i){
        for (unsigned j = 0;j < w; ++j){
            std::cout << table[i * h + j]  << ' ';
        }

        std::cout << std::endl;
    }
}

png::image<png::rgb_pixel> create_image(unsigned w, unsigned h, unsigned *table){

    printf("w = %d, h = %d\n",w,h);
    png::image< png::rgb_pixel > image(w, h);

    #pragma omp parallel for
    for (png::uint_32 y = 0; y < image.get_height(); ++y)
    {
        for (png::uint_32 x = 0; x < image.get_width(); ++x)
        {
            if (table[y * w + x] == 0){
                image[y][x] = png::rgb_pixel(30, 30, 30);
            }
            else{
                image[y][x] = png::rgb_pixel(table[y * w + x] * 2, table[y * w + x] * 2, 170 + table[y * w + x] * 2);
            }
        }
    }

    return image;
}


struct params {
    COMPLEX::complex<REAL_TYPE> c0,c1;
    unsigned w,h,n_threads;
    mandelbrot::exec_mode ex;
    std::string output_path;

    params(
        const COMPLEX::complex<REAL_TYPE> &c0, const COMPLEX::complex<REAL_TYPE> &c1,
        unsigned w, unsigned h, unsigned n_threads,
        mandelbrot::exec_mode ex, const std::string &output_path
    ): c0(c0), c1(c1), w(w), h(h), n_threads(n_threads), ex(ex), output_path(output_path)
    {}
};

mandelbrot::exec_mode get_exec_mode(const char * mode){
    mandelbrot::exec_mode ex;
    if (mode[0] == '0'){
        ex = mandelbrot::exec_mode::CPU;
    }
    else if (mode[0] == '1'){
        if (!__CUDA__){
            std::cerr << "WARNING! You chose to use GPU execution without using nvcc" << std::endl;
            std::cerr << "\tDefaulting to CPU execution..." << std::endl;
            return mandelbrot::exec_mode::CPU;
        }
        ex = mandelbrot::exec_mode::GPU;
    }
    else{
        std::cerr << "Invalid execution mode (0 or 1 is allowed)!" << std::endl;
        exit(1);
    }

    return ex;
}

struct params parse_args(int argc, char **argv){

    using COMPLEX::complex;


    std::string
        usage("USAGE: dmbrot <C0_REAL_TYPE> <C0_IMAG> <C1_REAL> <C1_IMAG> <W> <H> <CPU/GPU> <THREADS> <OUTPUT>");

    if (argc != 10){
        std::cerr << usage << std::endl;
        exit(1);
    }

    REAL_TYPE
        c0_real = atof(argv[1]), c0_imag = atof(argv[2]),
        c1_real = atof(argv[3]), c1_imag = atof(argv[4]);

    unsigned w = atoi(argv[5]), h = atoi(argv[6]), n_threads = atoi(argv[8]);

    const complex<REAL_TYPE> c0(c0_real, c0_imag), c1(c1_real, c1_imag);

    std::cout << c0 << ' ' << c1 << ' ' << w << ' ' << h << std::endl;

    return params(
        c0,c1,
        w, h, n_threads, get_exec_mode(argv[7]),argv[9]
    );
}


int main(int argc, char **argv){
    using COMPLEX::complex;

    MPI_Init(&argc, &argv);

    params args = parse_args(argc, argv);

    const mandelbrot::exec_mode ex = args.ex;
    const unsigned w = args.w, h = args.h, m = 250;

    unsigned *table = new unsigned[w * h];

    complex<REAL_TYPE> c0(args.c0),c1(args.c1);

    const REAL_TYPE delta_x = (c1.real() - c0.real()) / w;
    const REAL_TYPE delta_y = (c1.imag() - c0.imag()) / h;

    int world_size, world_rank;

    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    if (world_rank == 0){
        unsigned block_size = mandelbrot::get_block_size(w, h, world_size - 1);

        for (unsigned i = 1; i < world_size; ++i){
            int *bound = mandelbrot::get_boundaries(w, h, i, world_size - 1, block_size);
            int start_x = bound[0], end_x = bound[1], start_y = bound[2], end_y = bound[3];
            unsigned local_h = end_y - start_y + 1;
            unsigned local_table_size = w * local_h;
            unsigned offset = block_size * w * (i - 1);

            printf("\t%d - start_y: %d, end_y: %d, start_x: %d, end_x: %d, block_size: %d, table_size: %d, offset: %d\n", i, start_y, end_y, start_x, end_x, block_size, local_table_size, offset);
            printf("\tRank: %d will receive %d!\n", i, local_table_size);
            MPI_Recv(
                table + offset, // TODO: Calc table offset according to rank
                local_table_size,
                MPI_UNSIGNED,
                i,
                0,
                MPI_COMM_WORLD,
                MPI_STATUS_IGNORE
            );
            printf("\tRank: %d received data!\n", i);
        }
            
        png::image< png::rgb_pixel > image = create_image(w,h,table);
        image.write(args.output_path);

    }
    else{
        d_mandelbrot(
            ex, args.n_threads,
            world_rank, world_size - 1,
            c0, c1,
            delta_x, delta_y,
            w, h, m, table
        );
    }

    delete[] table;

    MPI_Finalize();

    return 0;
}
